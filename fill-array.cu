#include <iostream>
#include <hip/hip_runtime.h>

__global__ void fill(float *dest, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        dest[i] = i + 1;
    }
}

int main() {
    int N = 200;
    size_t size = N * sizeof(float);

    float *h_dest = (float*)malloc(size);

    float *d_dest;
    hipMalloc(&d_dest, size);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    fill<<<gridSize, blockSize>>>(d_dest, N);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA kernel failed: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    hipMemcpy(h_dest, d_dest, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << h_dest[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_dest);
    free(h_dest);

    return 0;
}
